#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#define N 100

/*
extern "C" __global__ void kernel(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    c[tid] = a[tid] + b[tid];
}
*/
int main()
{   
    
    /*int a[N], b[N], c[N];
    CUdeviceptr d_a, d_b, d_c;

    for (int i = 0; i < N; ++i)
    {
        a[i] = N - i;
        b[i] = i * i;
    }*/

    printf("*************cuda_init*************\n");
    hipInit(0);
    /*printf("*************cuda_device_get*************\n");
    CUdevice device;
    cuDeviceGet(&device, 0);

    printf("*************cuda_create_contex*************\n");
    CUcontext context;
    cuCtxCreate(&context, 0, device);
    printf("*************cuda_module_load*************\n");
    CUmodule module;
    const char *module_file = "kernel.ptx";
    cuModuleLoad(&module, module_file);

    CUfunction function;
    const char *kernel_name = "kernel";
    cuModuleGetFunction(&function, module, kernel_name);
    printf("*************cuda_malloc_1*************\n");
    cuMemAlloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    cuMemAlloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    cuMemAlloc(&d_c, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_1*************\n");
    cuMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    cuMemcpyHtoD(d_b, b, sizeof(int) * N);

    // Launch the kernel
    void *args[3] = {&d_a, &d_b, &d_c};
    printf("*************cuda_LacunhKernel*************\n");
    raise(SIGTRAP);
    cuLaunchKernel(function, N, 1, 1, 1, 1, 1, 0, 0, args, 0);

    // Copy result back to host
    printf("*************cuda_memcpyDtoh*************\n");
    cuMemcpyDtoH(c, d_c, sizeof(int) * N);

    // Free device memory
    printf("*************cuda_Free_1*************\n");
    cuMemFree(d_a);
    printf("*************cuda_Free_2*************\n");
    cuMemFree(d_b);
    printf("*************cuda_Free_3*************\n");
    cuMemFree(d_c);

    // Cleanup CUDA
    printf("*************cuCtxDestroy*************\n");
    cuCtxDestroy(context);*/
    return 0;
}

/*dekstametazon neomicin kapi*/