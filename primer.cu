#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"
#include "structs.h"

#define N 512
int main()
{   
    int a[N], b[N],c[N],control[N];
    hipDeviceptr_t d_a; //, d_b, d_c;
    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i ;control[i] = i + i;}

    //map(getpid());

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");

    //mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);

    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);

    ///uint64_t *ptr = (uint64_t*)0x7ffff7fab000;
    //map(getpid());
    //for(uint64_t *ptr = (uint64_t*)0x7ffff7fa2000; ptr <(uint64_t*)0x7ffff7fb2000; ptr ++){ if (*ptr){ printf("%p %lx \n", ptr, *ptr); }}
    
    //map(getpid());    

    /*
		printf("*************cuda_create_contex_start*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    printf("*************cuda_create_contex_ende*************\n");
    */

    exit(0);
    
    printf("*************cuda_malloc_1*************\n"); // cini se da mallloc poziva samo je 
    hipMalloc(&d_a, sizeof(int) * N); // ovo je 5c000091 objekat
    mprotect((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000 , PROT_READ | PROT_WRITE);

		printf("d_a = %p  %p \n" ,(uint64_t*)d_a , &d_a );

    exit(1);

    munmap((void*)0x7fffea000000, 0x7ffff0000000-0x7fffea000000);
    munmap((void*)0x7fffce400000, 0x7fffce600000-0x7fffce400000);
    munmap((void*)0x7fffcf000000, 0x7fffcf200000-0x7fffcf000000);
    munmap((void*)0x7fffcf400000, 0x7fffcf577000-0x7fffcf400000);
    munmap((void*)0x7fffe2fe0000, 0x7fffe2fe1000-0x7fffe2fe0000);
    munmap((void*)0x7fffe2fe1000, 0x7fffe2fe2000-0x7fffe2fe1000);
    munmap((void*)0x7fffe2fe2000, 0x7fffe2fe3000-0x7fffe2fe2000);
    munmap((void*)0x7fffe2fe3000, 0x7fffe2fe4000-0x7fffe2fe3000);
    munmap((void*)0x7fffe2fe4000, 0x7fffe2fe5000-0x7fffe2fe4000);
    munmap((void*)0x7fffe2fe5000, 0x7fffe2fe6000-0x7fffe2fe5000);
    munmap((void*)0x7fffe2fe6000, 0x7fffe2fe7000-0x7fffe2fe6000);
    munmap((void*)0x7fffe2fe8000, 0x7fffe2fe9000-0x7fffe2fe8000);
    munmap((void*)0x7fffe2fe9000, 0x7fffe2fea000-0x7fffe2fe9000);
    munmap((void*)0x7fffe2fea000, 0x7fffe2feb000-0x7fffe2fea000);
    munmap((void*)0x7fffe2fec000, 0x7fffe2fed000-0x7fffe2fec000);
    munmap((void*)0x7fffe2fed000, 0x7fffe2fee000-0x7fffe2fed000);
    munmap((void*)0x7fffe2fee000, 0x7fffe2fef000-0x7fffe2fee000);
    munmap((void*)0x7fffe2ff0000, 0x7fffe2ff1000-0x7fffe2ff0000);
    munmap((void*)0x7fffe2ff1000, 0x7fffe2ff2000-0x7fffe2ff1000);
    munmap((void*)0x7fffe2ff2000, 0x7fffe2ff3000-0x7fffe2ff2000);
    /*
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);
    */

    //dev/nvidia0
    //dev/nvidia-uvm
    //munmap((void*)0x205000000 , 0x205200000-0x205000000);   // ne

    //dev/nvidiactl 
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x200000000, 0x200200000-0x200000000);
    munmap((void*)0x204c00000, 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000, 0x205000000-0x204e00000);   // ne
    //munmap((void*)0x205200000, 0x205400000-0x205200000);   // ne

    /*hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);*/


    //munmap((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000);
    //memset((void*)c , 0x0 ,512);
    
    clear_nvctrl();
    printf("*************cuCopyHosttoDevice*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int)*N); 

 
    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_a, sizeof(int) * N);
    hexdump((void*)c , 0x10);

    //map(getpid());

    // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
		
		// 200200000-200400000 rw-s 00000000 00:05 1031                             /dev/nvidia0
    printf("CONTEXXXXXXXXXXXXX FREEEEEEEEEEEEEEEEEEEEEEEEEEEEEe\n");
    //hipCtxDestroy(context);

    return 0;
}
// p $_siginfo._sifields._sigfault.si_addr
/*
izgleda da cudainit pravi 5c000002 i mozda  5c000003
dok cuda contex mappira sranja
*/

/*
0x7fffe2feb010 : 1550 
0x7fffe2feb014 : 74e4c9c0 
0x7fffe2feb018 : 90017        #door bell value
0x7fffe2feb01c : ffff0000 

*/