#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"

#define N 10
int main()
{   
    int a[N], b[N],c[N],control[N];
    hipDeviceptr_t d_a, d_b, d_c , d_d;

    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i;}

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");
    mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);

    hipDevice_t device;
    hipDeviceGet(&device , 0);
    printf("DEVICE = %p -> %x \n" , &device , device);

		printf("*************cuda_create_contex_start*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    printf("*************cuda_create_contex_ende*************\n");
    mprotect((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000 , PROT_READ | PROT_WRITE);


    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);
    uint64_t *door_bell = (uint64_t*)0x7ffff7fab090;
    hipMalloc(&d_d, sizeof(int) * 0x1000000);
  

    printf("*************cuCopyHosttoDevice*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int)*N);
    hipMemcpyHtoD(d_b, b, sizeof(int)*N);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    hipFunction_t function;
    printf("da = %lx db = %lx dc = %lx\n" ,(uint64_t)d_a , (uint64_t)d_b ,(uint64_t)d_c);

    hipMemcpyDtoH(b, d_a, sizeof(int) * N);

    int res = hipModuleLoad(&module, "kernel.cubin");
    printf("res = %x\n" , res);
    assert(res == 0);



    map(getpid()); // 0x205607000
    exit(1);
    

    munmap((void*)0x7fffcf577000 , 0x7fffe0000000-0x7fffcf577000);
    munmap((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000);

    //7fffcfe3b300 ovo je programska adresa, nema nista 0x7fffcf577000 <= 0x7fffcfe3b300 <= 0x7fffe0000000 ovo  odmapiram i sve radi
    //7fffce220000 7fffcc000000-7fffce400000 , ovde  nema nista ? 
    //7fffce010000 7fffcc000000-7fffce400000 , ovde  nema nista ?
    //7fffeb000000  7fffea000000-7ffff0000000 ---p 00000000 00:00 0 nema nista 
    //7fffed000000  7fffea000000-7ffff0000000
    //7fffce220160  7fffea000000-7ffff0000000 7fffcc000000-7fffce400000 
    
    printf("*************FUNCTION*************\n");
    int res_2 = hipModuleGetFunction(&function, module, "kernel"); assert(res_2 == 0);  // p $_siginfo._sifields._sigfault.si_addr


    clear_nvctrl();
    munmap((void*)205600000 , 205800000-205600000);
    //munmap((void*)205400000 , 205600000-205400000);

    printf("*************KERNEL_LAUNNCH*************\n");
    void *args[3] = { &d_a, &d_a, &d_c};
    int res_3 = hipModuleLaunchKernel(function, 1, 1, 1, N, 1, 1, 0, 0, args, 0);
    assert(res_3 == 0);
    //dump_small((void*)0x200400000 ,(void*)0x203c00000);

    printf("*************DEVICE_TO_HOST*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);

    dump_small((void*)0x200400000 ,(void*)0x203c00000);
    hexdump((void*)c , 0x20);
    //for(int i  = 0 ; i < N ; i++){printf("%x " ,c[i]);}printf("\n");
    return 0;
}


//dump_small((void*)0x200400000 ,(void*)0x203c00000);
//7fffcc000000-7fffce400000  ove adrese su zanimljive ovde se nalazi offset out  upper, ovde nema nista 
// 7fff ce220000
// 7fff ce220160 sta je ovo
// 7fff ce221860