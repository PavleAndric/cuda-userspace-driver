#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"
#include "structs.h"

#define N 512
int main()
{   
    int a[N], b[N],c[N],control[N];
    hipDeviceptr_t d_a; //, d_b, d_c;
    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i ;control[i] = i + i;}

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");

    mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);

    //map(getpid());
    //for(uint32_t *ptr = (uint32_t*)0x200000000 ; ptr <(uint32_t*)0x300200000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}
    //for(uint32_t *ptr = (uint32_t*)0x7ffff5600000 ; ptr <(uint32_t*)0x7ffff5e00000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}
    
    hipDevice_t device;
    hipDeviceGet(&device , 0);
    printf("&device = %p device = %x\n" , &device , device);
    
		printf("*************cuda_create_contex_start*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    printf("*************cuda_create_contex_ende*************\n");
    
    mprotect((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000 , PROT_READ | PROT_WRITE);

    printf("*************cuda_malloc_1*************\n"); // izglead da za malloc  ne treba BAR adresa, ne slama se akd se odmapira BAR
    hipMalloc(&d_a, sizeof(int) * N); // ovo je 5c000091 objekat 
		printf("d_a = %p  %p \n" ,(uint64_t*)d_a , &d_a );


    munmap((void*)0x7fffea000000, 0x7ffff0000000-0x7fffea000000);
    munmap((void*)0x7fffce400000, 0x7fffce600000-0x7fffce400000);
    munmap((void*)0x7fffcf000000, 0x7fffcf200000-0x7fffcf000000);
    munmap((void*)0x7fffcf400000, 0x7fffcf577000-0x7fffcf400000);
    munmap((void*)0x7fffe2fe0000, 0x7fffe2fe1000-0x7fffe2fe0000);
    munmap((void*)0x7fffe2fe1000, 0x7fffe2fe2000-0x7fffe2fe1000);
    munmap((void*)0x7fffe2fe2000, 0x7fffe2fe3000-0x7fffe2fe2000);
    munmap((void*)0x7fffe2fe3000, 0x7fffe2fe4000-0x7fffe2fe3000);
    munmap((void*)0x7fffe2fe4000, 0x7fffe2fe5000-0x7fffe2fe4000);
    munmap((void*)0x7fffe2fe5000, 0x7fffe2fe6000-0x7fffe2fe5000);
    munmap((void*)0x7fffe2fe6000, 0x7fffe2fe7000-0x7fffe2fe6000);
    munmap((void*)0x7fffe2fe8000, 0x7fffe2fe9000-0x7fffe2fe8000);
    munmap((void*)0x7fffe2fe9000, 0x7fffe2fea000-0x7fffe2fe9000);
    munmap((void*)0x7fffe2fea000, 0x7fffe2feb000-0x7fffe2fea000);
    munmap((void*)0x7fffe2fec000, 0x7fffe2fed000-0x7fffe2fec000);
    munmap((void*)0x7fffe2fed000, 0x7fffe2fee000-0x7fffe2fed000);
    munmap((void*)0x7fffe2fee000, 0x7fffe2fef000-0x7fffe2fee000);
    munmap((void*)0x7fffe2ff0000, 0x7fffe2ff1000-0x7fffe2ff0000);
    munmap((void*)0x7fffe2ff1000, 0x7fffe2ff2000-0x7fffe2ff1000);
    munmap((void*)0x7fffe2ff2000, 0x7fffe2ff3000-0x7fffe2ff2000);
    /*
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);
    */

    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x200000000, 0x200200000-0x200000000);
    munmap((void*)0x205200000, 0x205400000-0x205200000);   // ne
    munmap((void*)0x205000000 , 0x205200000-0x205000000);  // ne
    munmap((void*)0x204c00000, 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000, 0x205000000-0x204e00000);   // ne


    //memset((void*)0x200200000 , 0x0 , 0x400000); // clear command buffer 
    //clear_nvctrl();
    
    //map(getpid());
    //CINI SE DA OVO NIJE BITNO
    //memset((void*)0x7fffe2fdf000 , 0x100, 0x1000);
    //memset((void*)0x7fffe2fe7000 , 0x100, 0x1000);
    //memset((void*)0x7fffe2feb000 , 0x100, 0x1000);
    //memset((void*)0x7fffe2fef000 , 0x100, 0x1000);

    //munmap((void*)0x7ffff7fab000 , 0x10000);
    printf("*************cuCopyHosttoDevice*************\n"); 
    hipMemcpyHtoD(d_a, a, sizeof(int)*N);

    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_a, sizeof(int) * N);
    printf("DUMP  C \n");
    hexdump((void*)c , 0x10);

    // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);

    //for(uint32_t *ptr = (uint32_t*)0x7fffe2fdf000 ; ptr <(uint32_t*)0x7fffe2fe0000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}
    //printf("DOOR_BELLS = %x %x %x %x \n" , *((uint32_t*)(0x7fffe2fdf000 + 0x18)) , *((uint32_t*)(0x7fffe2fe7000 + 0x18)) ,  *((uint32_t*)(0x7fffe2feb000 + 0x18)) , *((uint32_t*)(0x7fffe2fef000 + 0x18)));
    //hipCtxDestroy(context);
    return 0;
}
/*

printf("CONTEXXXXXXXXXXXXX FREEEEEEEEEEEEEEEEEEEEEEEEEEEEEe\n");

p $_siginfo._sifields._sigfault.si_addr
for(uint32_t *ptr = (uint32_t*)0x7fffe2fdf000 ; ptr <(uint32_t*)0x7fffe2fe0000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}} // odredjene BAR adrese

7fffe2fdf000-7fffe2fe0000 rw-s 00000000 00:05 1029 0x1000
7fffe2fe7000-7fffe2fe8000 rw-s 00000000 00:05 1029 0x1000
7fffe2feb000-7fffe2fec000 rw-s 00000000 00:05 1029 0x1000
7fffe2fef000-7fffe2ff0000 rw-s 00000000 00:05 1029 0x1000

Za 5c000016:
  alokacija objekta
  NV_ESC_RM_MAP_MEMORY objekta
  parvi se c46f
  NV_ESC_RM_CONTROL 5c000002 NV0080_CTRL_FIFO_GET_CHANNELLIST_PARAMS
  NV0000_CTRL_CLIENT_GET_ADDR_SPACE_TYPE_PARAMS
  parvi se c5c0 koje je c46f parent
  NV906F_CTRL_GET_CLASS_ENGINEID_PARAMS c5c0
  parvi se c5b5 kome je c46f parent
  NV906F_CTRL_GET_CLASS_ENGINEID_PARAMS
  c5c0 poziva  NVC36F_CTRL_CMD_GPFIFO_GET_WORK_SUBMIT_TOKEN_PARAMS
  sub device pozvia NV2080_CTRL_GR_GET_CTX_BUFFER_SIZE_PARAMS gde je c5c0 arg 
  KRAJ


svu su deca 5c000002 svi nvidia ctrl fd
// 7fffe2fdf000-7fffe2fe0000
PRVI NV_ESC_RM_MAP_MEMORY, fd=10 hDevice=5c000002, len=1000, offset=0, flags=c0000, linaddr=(nil), hmem_=5c000016,hClient=c1d04d15, status__=0 
//7fffe2fe7000-7fffe2fe8000
DRUGI NV_ESC_RM_MAP_MEMORY, fd=12 hDevice=5c000002, len=1000, offset=0, flags=c0000, linaddr=(nil), hmem_=5c00003c,hClient=c1d04d15, status__=0 
//7fffe2feb000-7fffe2fec000
TRECI NV_ESC_RM_MAP_MEMORY, fd=14 hDevice=5c000002, len=1000, offset=0, flags=c0000, linaddr=(nil), hmem_=5c00004a,hClient=c1d04d15, status__=0
//7fffe2fef000-7fffe2ff0000
CETVRTI NV_ESC_RM_MAP_MEMORY, fd=16 hDevice=5c000002, len=1000, offset=0, flags=c0000, linaddr=(nil), hmem_=5c000058,hClient=c1d04d15, status__=0 
*/
/*
0x7fffe2feb010 : 1550 
0x7fffe2feb014 : 74e4c9c0 
0x7fffe2feb018 : 90017        #door bell value
0x7fffe2feb01c : ffff0000 
*/