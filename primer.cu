#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"
#define N 512
int main()
{   
    int a[N], b[N],control[N];
    int c[N];
    hipDeviceptr_t d_a, d_b, d_c;
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
        b[i] = i;
        control[i] = i + i;
    }

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");

    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    printf("*************cuda_create_contex*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);

    hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);

    //dev/nvidia0
    // dev/nvidia-uvm
    munmap((void*)0x205000000 , 0x205200000-0x205000000);   // ne

    // /dev/nvidiactl 
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x204c00000 , 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000 , 0x205000000-0x204e00000);   // ne
    munmap((void*)0x205200000 , 0x205400000-0x205200000);   // ne

		mprotect((void*)0x200000000, 0x200200000-0x200000000, PROT_READ);

    
    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    //printf("DESINGER %llx %p\n" , d_a , &d_a);
    printf("*************cuda_memcpyHtod_1*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    hipMemcpyHtoD(d_b, b, sizeof(int) * N);

    //unsigned int * romcina = (unsigned int *)0xB0000B;
    printf("_____%llx_____  \n" ,d_a);
    printf("_____%llx_____  \n" ,d_b);
    printf("_____%llx_____  \n" ,d_c);
    void *args[3] = {&d_a, &d_b, &d_c};
  
    printf("*************cuda_LacunhKernel*************\n");
    munmap((void*)0x200436000 , 0x202c00000-0x200436000); 
    munmap((void*)0x200400000 , 0x200435000-0x200400000); 
    munmap((void*)0x200230000, 0x200400000-0x200230000);
    uint64_t rom = gas();
    uint64_t *ptr = (uint64_t*)rom;
    uint64_t *ptr_2 = (uint64_t*)((uint64_t)ptr + 0x10000);
    uint64_t *ptr_3 = (uint64_t*)((uint64_t)ptr + 0x90);
    printf("NVIDIA0 END | START %p-%p , SEG: %p , %x\n" , ptr ,ptr_2 , ptr_3, *ptr_3);

    // END | START 0x7ffff7fab000-0x7ffff7fbb000 SEG 0x7ffff7fab090
    /*while (ptr < ptr_2){
      if (*ptr != 0){printf("%p:%lx\n" , ptr , *ptr);}
      ptr++; 
    }*/
    int gas = munmap((void*)(rom+0x1000) , (uint64_t)ptr_2-(rom+0x1000));printf("unmap=%d \n" , gas); // 0x10000  //17ad09 0e8bf37de0,  17ad09 1304ea0000 , 17ad09 157dfdedc0
    hipModuleLaunchKernel(function, 32, 16, 1, 1, 1, 1, 0, 0, args, 0);
    printf("SEG AFTER %p %x \n" , ptr_3, *ptr_3);

    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);
    for(int i = 0 ; i < N ; i ++){ assert(c[i] == control[i]); }

    /* // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
    printf("*************cuda_Free_2*************\n");
    hipFree(d_b);
    printf("*************cuda_Free_3*************\n");
    hipFree(d_c);
		printf(":D :D :D\n"); */
    
    return 0;
}
//  p $_siginfo._sifields._sigfault.si_addr