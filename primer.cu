#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>

#define N 100

void dump(){
	// dump nvidia0 0x200200000-0x200400000
	printf("IDE_GAS\n");
	uint32_t *ptr = (uint32_t*)0x200200000;
	printf("/dev/nvidia0\n");
	while (ptr != (uint32_t*)0x200400000){ 
		if (*ptr != 0){
			printf("%p %x\n ",ptr ,*ptr); 
		}
		ptr ++;
	}
}
int main()
{   
    int a[N], b[N], c[N] ,control[N];
    hipDeviceptr_t d_a, d_b, d_c;
    
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
        b[i] = i;
        control[i] = i + i;
    }
    char buf[1500];
    pid_t pid = getpid();
    char path[100];

    sprintf(path , "/proc/%d/maps" , pid);

    printf("*************cuda_init*************\n");
    hipInit(0); // MMAP 0x200000000-300200000 uvek ! ! ! 
    printf("*************end_cudainit*************\n");

    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    printf("*************cuda_create_contex*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);

    hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);

		/*
    //dev/nvidia0
    munmap((void*)0x200200000 , 0x200000); mora            

    // /dev/nvidiactl 
    munmap((void*)0x200400000 , 0x203c00000-0x200400000);   mora
    munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   mora
    munmap((void*)0x205600000 , 0x205800000-0x205600000);   mora
    munmap((void*)0x204c00000 , 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000 , 0x205000000-0x204e00000);   // ne
    munmap((void*)0x205200000 , 0x205400000-0x205200000);   // ne*/
			
		//mprotect((void*)0x200200000 ,0x200400000-0x200200000, PROT_READ);
    // dump nvidia0 0x200200000-0x200400000

		//mprotect((void*)0x204600000, 0x204800000-0x204600000, PROT_READ);

    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    printf("*************cuda_memcpyHtod_1*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    hipMemcpyHtoD(d_b, b, sizeof(int) * N);
    
    void *args[3] = {&d_a, &d_b, &d_c};
    printf("*************cuda_LacunhKernel*************\n");
    hipModuleLaunchKernel(function, N, 1, 1, 1, 1, 1, 0, 0, args, 0);
    
    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);
    for(int i = 0 ; i < N ; i ++){assert(c[i] == control[i]);}
    
    // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
    printf("*************cuda_Free_2*************\n");
    hipFree(d_b);
    printf("*************cuda_Free_3*************\n");
    hipFree(d_c);

		printf(":D :D :D\n");
    FILE *f = fopen(path, "r");
		while (fgets(buf, sizeof(buf), f) != NULL){printf("%s", buf);}
		printf("\n");
		fclose(f);
		
		dump();
    return 0;
}

/*
0x7f719c7de000 /dev/zero
0x7f719c7de000 /dev/nvidiactl
0x205000000 /dev/nvidia-uvm
0x205000000 /dev/nvidia-uvm
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f719c5de000 /dev/zero
0x7f719c5de000 /dev/zero
0x205a00000 /dev/zero
0x205a00000 /dev/zero
0x205a00000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
*/