#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include <inttypes.h>

#define N 124

/*
200000000-200200000 r--p 00000000 00:00 0 
200200000-200400000 rw-s 00000000 00:05 1020                             /dev/nvidia0
200400000-203c00000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
203c00000-204a00000 ---p 00000000 00:00 0 
204a00000-204c00000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
205400000-205600000 ---p 00000000 00:00 0 
205600000-205800000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
205800000-205a00000 ---p 00000000 00:00 0 
205a00000-205c00000 rw-s 00000000 00:01 180536                           /dev/zero (deleted)
*/

void dump_large(){

  uint32_t *base = (uint32_t*)0x200200000;

  while (base != (uint32_t*)0x200400000){ // 0x200400000 // 0x205800000 OVO
    //if (base != NULL)
    if (base == (uint32_t*)0x200400000){ printf("/dev/nvidiact 1 \n");}
    if (base == (uint32_t*)0x204c00000){ printf("/dev/nvidiact 3 \n"); base = (uint32_t*)0x205600000;}
    if (base == (uint32_t*)0x203c00000){ printf("/dev/nvidiact 2 \n"); base = (uint32_t*)0x204a00000;}
    if (*base != 0){
      uint32_t *bonus = (uint32_t*)(0x200000000 + *base);
      printf("ptr: (%p %x) bonus(%p" , base, *base , bonus);
      if (*bonus != 0) {printf(" %x" , *bonus);}
      printf(")\n");
    } 
    base ++;
  }
}
int main()
{   
    int a[N], b[N], c[N] ,control[N];
    hipDeviceptr_t d_a, d_b, d_c;
    //unsigned long long cigan = 0xB00B00; 
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
        b[i] = i;
        control[i] = i + i;
    }
    char buf[1500];
    pid_t pid = getpid();
    char path[100];

    sprintf(path , "/proc/%d/maps" , pid);

    printf("*************cuda_init*************\n");
    hipInit(0); // MMAP 0x200000000-300200000 uvek ! ! ! 
    printf("*************end_cudainit*************\n");

    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    printf("*************cuda_create_contex*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);

    hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);

    //dev/nvidia0
    //munmap((void*)0x200200000 , 0x200000); mora            

    // dev/nvidia-uvm
    munmap((void*)0x205000000 , 0x205200000-0x205000000);   // ne

    // /dev/nvidiactl 
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x204c00000 , 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000 , 0x205000000-0x204e00000);   // ne
    munmap((void*)0x205200000 , 0x205400000-0x205200000);   // ne

    // ???
    //munmap((void*)0x205c00000 , 0x300200000-0x205c00000);   // ne
    //munmap((void*)0x200000000 , 0x200200000-0x200000000);   // ne

		//mprotect((void*)0x204600000, 0x204800000-0x204600000, PROT_READ);
		mprotect((void*)0x200000000, 0x200200000-0x200000000, PROT_READ); // init

    
    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    //printf("DESINGER %llx %p\n" , d_a , &d_a);
    printf("*************cuda_memcpyHtod_1*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    hipMemcpyHtoD(d_b, b, sizeof(int) * N);

    //unsigned int * romcina = (unsigned int *)0xB0000B;
    void *lmao = (void*)(0xb00000b);
    printf("_____%llx_____  \n" ,d_a);
    printf("_____%llx_____  \n" ,d_b);
    printf("_____%llx_____  \n" ,d_c);
    //sleep(1000000);
    void *args[4] = {&d_a, &d_b, &d_c};
  
    printf("*************cuda_LacunhKernel*************\n");
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   //  0x205607f7c seg fault
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   //  0x204a0fff0 seg fault ptr:(0x204a0fff0 69)
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);     //  0x200434424

    hipModuleLaunchKernel(function, N, 1, 1, 1, 1, 1, 0, 0, args, 0); 
    /*
    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);
    for(int i = 0 ; i < N ; i ++){assert(c[i] == control[i]);}
    // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
    printf("*************cuda_Free_2*************\n");
    hipFree(d_b);
    printf("*************cuda_Free_3*************\n");
    hipFree(d_c);
    
		printf(":D :D :D\n");
    FILE *f = fopen(path, "r");
		while (fgets(buf, sizeof(buf), f) != NULL){printf("%s", buf);}
		printf("\n");
		fclose(f);
    //dump_large();

    printf("***** read\n");
    */
    uint32_t *ptr = (uint32_t*)0x200400000;
    while (ptr != (uint32_t*)0x203c00000) { if (*ptr != 0) printf("%p: %8x\n", ptr, *ptr); ++ptr; }
    
    return 0;
}

/*
0x7f719c7de000 /dev/zero
0x7f719c7de000 /dev/nvidiactl
0x205000000 /dev/nvidia-uvm
0x205000000 /dev/nvidia-uvm
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f719c5de000 /dev/zero
0x7f719c5de000 /dev/zero
0x205a00000 /dev/zero
0x205a00000 /dev/zero
0x205a00000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
*/