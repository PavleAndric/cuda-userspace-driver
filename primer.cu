#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include <inttypes.h>
#include "clc597.h"
#include "clc5c0.h"
#define N 512

//0x2004349c4:     hex(298) -> 12a
//0x200434bfc:     hex(298) -> 12a

// #define N 615
//0x2004353ac:     hex(615) -> 267
//0x2004355e4:     hex(615) -> 267

/*
200000000-200200000 r--p 00000000 00:00 0 
200200000-200400000 rw-s 00000000 00:05 1020                             /dev/nvidia0
200400000-203c00000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
203c00000-204a00000 ---p 00000000 00:00 0 
204a00000-204c00000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
205400000-205600000 ---p 00000000 00:00 0 
205600000-205800000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
205800000-205a00000 ---p 00000000 00:00 0 
205a00000-205c00000 rw-s 00000000 00:01 180536                           /dev/zero (deleted)
*/

// ovi su cudni
/*0x200400260: 2001255d | type:20000000  size:1  subc:1 mthd:174 
0x200400268: 2001255e | type:20000000  size:1  subc:1 mthd:178 
0x200400270: 2001255f | type:20000000  size:1  subc:1 mthd:17C 
0x200400278: 20012557 | type:20000000  size:1  subc:1 mthd:15C 
0x200400280: 20012558 | type:20000000  size:1  subc:1 mthd:160 
0x200400288: 20012559 | type:20000000  size:1  subc:1 mthd:164 */

void dump_small(){

  //200400000-203c00000 rw-s 00000000 00:05 1019                             /dev/nvidiactl
  //200400000-200436000 ovo je opseg u kojem zivi funkcija, moze  munmap 
  // 0x200433f84-0x2004354e4
  // 0x200433000-0x200436000
  // 0x200435000-0x200436000
  uint32_t *ptr = (uint32_t*)0x200435214; // 0x200433f84 ovde se pojavljuje prvi argument funckije 
  while (ptr != (uint32_t*)0x200436000) { // 0x2004354e4 ,ovo je opseg u kome sve radi
     if (*ptr != 0){
    
      int ide_gas = *ptr;
      int type =  ide_gas & 0xF0000000;
      int size = (ide_gas & 0x0FFF0000) >> 16;
      int subc = (ide_gas & 0x0000FF00) >> 13;
      int mthd = (ide_gas & 0x000000FF) << 2;
      if ((*ptr > 0x20000000 && *ptr < 0x30000000) ||(*ptr > 0x60000000 && *ptr < 0x70000000) ){
        printf("%p: %8x | type:%X  size:%X  subc:%X mthd:%X " ,ptr, *ptr, type, size, subc , mthd);
        //NVC5C0
        if (mthd == NVC5C0_SET_OBJECT) {printf("NVC5C0_SET_OBJECT\n");}
        else if  (mthd == NVC5C0_NO_OPERATION) {printf("NVC5C0_NO_OPERATION\n");}
        else if  (mthd == NVC5C0_SET_SHADER_SHARED_MEMORY_WINDOW_A) {printf("NVC5C0_SET_SHADER_SHARED_MEMORY_WINDOW_A\n");}
        else if  (mthd == NVC5C0_SET_SHADER_SHARED_MEMORY_WINDOW_B) {printf("NVC5C0_SET_SHADER_SHARED_MEMORY_WINDOW_B\n");}
        else if  (mthd == NVC5C0_SET_SPA_VERSION) {printf("NVC5C0_SET_SPA_VERSION\n");}
        else if  (mthd == NVC5C0_SET_CWD_REF_COUNTER) {printf("NVC5C0_SET_CWD_REF_COUNTER\n");}
        else if  (mthd == NVC5C0_SET_RESERVED_SW_METHOD07) {printf("NVC5C0_SET_RESERVED_SW_METHOD07\n");}
        else if  (mthd == NVC5C0_SET_RESERVED_SW_METHOD10) {printf("NVC5C0_SET_RESERVED_SW_METHOD10\n");}
        else if  (mthd == NVC5C0_SET_RESERVED_SW_METHOD09) {printf("NVC5C0_SET_RESERVED_SW_METHOD09\n");}
        else if  (mthd == NVC5C0_SET_VALID_SPAN_OVERFLOW_AREA_A) {printf("NVC5C0_SET_VALID_SPAN_OVERFLOW_AREA_A\n");}
        else if  (mthd == NVC5C0_SET_SHADER_LOCAL_MEMORY_NON_THROTTLED_C) {printf("NVC5C0_SET_SHADER_LOCAL_MEMORY_NON_THROTTLED_C\n");}
        else if  (mthd == NVC5C0_INVALIDATE_SKED_CACHES) {printf("NVC5C0_INVALIDATE_SKED_CACHES\n");}
        else if  (mthd == NVC5C0_LAUNCH_DMA) {printf("NVC5C0_LAUNCH_DMA\n");}
        else if  (mthd == NVC5C0_OFFSET_OUT_UPPER) { printf("NVC5C0_OFFSET_OUT_UPPER\n");}
        else if  (mthd == NVC5C0_LINE_LENGTH_IN) {printf("NVC5C0_LINE_LENGTH_IN\n");}
        else if  (mthd == NVC5C0_SET_SHADER_LOCAL_MEMORY_NON_THROTTLED_A) {printf("NVC5C0_SET_SHADER_LOCAL_MEMORY_NON_THROTTLED_A\n");}
        else if  (mthd == NVC5C0_SET_SHADER_LOCAL_MEMORY_NON_THROTTLED_B) {printf("NVC5C0_SET_SHADER_LOCAL_MEMORY_NON_THROTTLED_B\n");}
        else if  (mthd == NVC5C0_SET_INLINE_QMD_ADDRESS_A) {printf("NVC5C0_SET_INLINE_QMD_ADDRESS_A\n");}
        // NVC597
        else if  (mthd == NVC597_SET_PS_OUTPUT_SAMPLE_MASK_USAGE) {printf("NVC597_SET_PS_OUTPUT_SAMPLE_MASK_USAGE\n");}
        else if  (mthd == NVC597_SET_STREAM_OUT_BUFFER_LOAD_WRITE_POINTER(0)) {printf("NVC597_SET_STREAM_OUT_BUFFER_LOAD_WRITE_POINTER\n");}
        else if  (mthd == NVC597_SET_L1_CONFIGURATION) {printf("NVC597_SET_L1_CONFIGURATION\n");}
        else if  (mthd == NVC597_SET_ROOT_TABLE_VISIBILITY(0)) {printf("NVC597_SET_L1_CONFIGURATION\n");}
        // TYPE 6
        else if  (mthd == NVC597_LOAD_INLINE_DATA) { printf("NVC597_LOAD_INLINE_DATA\n");}
      }
      for (int j = 0 ; j < size; j ++){
        ptr++;
        if (j != 0 && j % 4 == 0){printf("\n");}
        printf("%08x " , *ptr); 
      }
      if (size != 0){
        printf("\n");
      }
    }
    ++ptr;
  } 
}
int main()
{   
    int a[N], b[N], c[N] ,control[N];
    hipDeviceptr_t d_a, d_b, d_c;
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
        b[i] = i;
        control[i] = i + i;
    }
    char buf[1500];
    pid_t pid = getpid();
    char path[100];

    sprintf(path , "/proc/%d/maps" , pid);

    printf("*************cuda_init*************\n");
    hipInit(0); // MMAP 0x200000000-300200000 uvek ! ! ! 
    printf("*************end_cudainit*************\n");

    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    printf("*************cuda_create_contex*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);

    hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);

    //dev/nvidia0
    //munmap((void*)0x200200000 , 0x200000); mora            

    // dev/nvidia-uvm
    munmap((void*)0x205000000 , 0x205200000-0x205000000);   // ne

    // /dev/nvidiactl 
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x204c00000 , 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000 , 0x205000000-0x204e00000);   // ne
    munmap((void*)0x205200000 , 0x205400000-0x205200000);   // ne

    // ???
    //munmap((void*)0x205c00000 , 0x300200000-0x205c00000);   // ne
    //munmap((void*)0x200000000 , 0x200200000-0x200000000);   // ne

		//mprotect((void*)0x204600000, 0x204800000-0x204600000, PROT_READ);
		mprotect((void*)0x200000000, 0x200200000-0x200000000, PROT_READ); // init

    
    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    //printf("DESINGER %llx %p\n" , d_a , &d_a);
    printf("*************cuda_memcpyHtod_1*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    hipMemcpyHtoD(d_b, b, sizeof(int) * N);

    //unsigned int * romcina = (unsigned int *)0xB0000B;
    printf("_____%llx_____%p  \n" ,d_a , &d_a);
    printf("_____%llx_____%p  \n" ,d_b, &d_b);
    printf("_____%llx_____%p  \n" ,d_c, &d_c);
    void *args[3] = {&d_a, &d_b, &d_c};
  
    printf("*************cuda_LacunhKernel*************\n");
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   //  0x205607f7c seg fault
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   //  0x204a0fff0 seg fault ptr:(0x204a0fff0 69)
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   //2  0x200434424
    
    // 200400000-200436000 rw-s 00000000 00:05 1019 /dev/nvidiactl OPSEG U KOJEM ZIVI KERNEL
    int gas = munmap((void*)0x200436000 , 0x202c00000-0x200436000); // je potrebno za kopuju 0x202c00000
    int gas_2 = munmap((void*)0x200400000 , 0x200435000-0x200400000); // 0x200433000-0x200400000 = 12288(decimal)
    // nvidia0
    int gas_3 = munmap((void*)0x200230000, 0x200400000-0x200230000);

    /*uint32_t *romBase = (uint32_t*)0x20043549c;
    uint32_t *rom1 = (uint32_t*)0x200435478;
    uint32_t *rom2 = (uint32_t*)0x20043548c;
    uint32_t *romXD = (uint32_t*)0x200435490;
    uint32_t *rom3 = (uint32_t*)0x20043549c;
    uint32_t *rom4 = (uint32_t*)0x2004354a8;
    uint32_t *rom5 = (uint32_t*)0x2004354ac;
    uint32_t *rom6 = (uint32_t*)0x2004354b0;
    uint32_t *rom7 = (uint32_t*)0x2004354b4;
    printf("%x %x %x %x %x %x %x %x %x \n", *romBase, *rom1, *rom2, *romXD, *rom3, *rom4, *rom5, *rom6, *rom7);*/
    
    // munmap((void*)0x7fc676400000, 0x7fc676600000-0x7fc676400000);

    //0x7fc676400000-0x7fc676600000
    hipModuleLaunchKernel(function, 16, 32, 1, 1, 1, 1, 0, 0, args, 0);
    uint32_t *rom = (uint32_t*)0x205607f7c;

    printf("%x lmao \n " , *rom);
    //hipModuleLaunchKernel(function, N, 1, 1, 1, 1, 1, 0, 0, args, 0); 
    printf("*************cuda_memcpyDtoh************* %p\n" , c);
    printf("%x \n " , *rom);
    hipMemcpyDtoH(c, d_c, sizeof(int) * N); // 0x202c00020 //for(int i = 0 ; i < N ; i ++){ assert(c[i] == control[i]); }
    dump_small();
    /*for(int i = 0 ; i < N ; i ++ ){
      printf("%d" , c[i]);
      if (i % 64 == 0 && i != 0){
        printf("\n");
      }
    } printf("\n");*/
    
    /* // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
    printf("*************cuda_Free_2*************\n");
    hipFree(d_b);
    printf("*************cuda_Free_3*************\n");
    hipFree(d_c);
		printf(":D :D :D\n"); */
    /*
    FILE *f = fopen(path, "r");
		while (fgets(buf, sizeof(buf), f) != NULL){printf("%s", buf);}
		printf("\n");
		fclose(f);*/
    return 0;
}

/*
0x7f719c7de000 /dev/zero
0x7f719c7de000 /dev/nvidiactl
0x205000000 /dev/nvidia-uvm
0x205000000 /dev/nvidia-uvm
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196600000 /dev/nvidiactl
0x7f7196600000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f7196800000 /dev/zero
0x7f719c5de000 /dev/zero
0x7f719c5de000 /dev/zero
0x205a00000 /dev/zero
0x205a00000 /dev/zero
0x205a00000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
0x7f7197200000 /dev/nvidiactl
0x7f7197200000 /dev/zero
*/