#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"

#define N 10
int main()
{   
    int a[N], b[N],c[N];
    hipDeviceptr_t d_a, d_b, d_c , d_d;

    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i;}

    hipInit(0);
    mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);

    hipDevice_t device;
    hipDeviceGet(&device , 0);

    hipCtx_t context;
    hipCtxCreate(&context, 0, device);

    hipMalloc(&d_a, sizeof(int) * N);
    hipMalloc(&d_b, sizeof(int) * N);
    hipMalloc(&d_c, sizeof(int) * N);

    hipMalloc(&d_d, sizeof(int) * 0x1000000);

    hipMemcpyHtoD(d_a, a, sizeof(int)*N);
    hipMemcpyHtoD(d_b, b, sizeof(int)*N);

    hipModule_t module;
    hipFunction_t function;
    printf("da = %lx db = %lx dc = %lx\n" ,(uint64_t)d_a , (uint64_t)d_b ,(uint64_t)d_c);

    hipMemcpyDtoH(b, d_a, sizeof(int) * N);

    int res = hipModuleLoad(&module, "kernel.cubin");
    printf("res = %x\n" , res);
    assert(res == 0);

    munmap((void*)0x7fffcf577000 , 0x7fffe0000000-0x7fffcf577000);
    munmap((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000);
    
    int res_2 = hipModuleGetFunction(&function, module, "kernel"); assert(res_2 == 0);
    clear_nvctrl();
    munmap((void*)205600000 , 205800000-205600000);

    void *args[3] = { &d_a, &d_a, &d_c};
    int res_3 = hipModuleLaunchKernel(function, 1, 1, 1, N, 1, 1, 0, 0, args, 0);
    assert(res_3 == 0);

    hipMemcpyDtoH(c, d_c, sizeof(int) * N);
    return 0;
}
//p $_siginfo._sifields._sigfault.si_addr
