#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"
#include "structs.h"

#define N 512
int main()
{   
    int a[N], b[N],c[N],control[N];
    hipDeviceptr_t d_a, d_b, d_c;
    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i;control[i] = i + i;}

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");

    mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);
    
    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);

    printf("*************cuda_create_contex_start*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    printf("*************cuda_create_contex_ende*************\n");
    
    //map(getpid());
    //dump_dumb((void*)0x200400000 ,(void*)0x203c00000);
    //dump_dumb((void*)0x7fffce400000 ,(void*)0x7fffe2ff3000);
    //dump_dumb((void*)0x7fffe2fdf000 ,(void*)0x7fffe2ff3000); // ovde nema nista
    //dump_dumb((void*)0x7fffce400000 ,(void*)0x7fffcea00000); // ovde nema nista
    //dump_dumb((void*)0x7fffcf000000 ,(void*)0x7fffcf577000); // ovde nema nista

    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);

    printf("d_a = %p  \n" ,(uint64_t*)d_a );

    dump_dumb((void*)0x7fffe2fef000 ,(void*)0x7fffe2ff0000);  

    // OVDE SE NALAZE VREDNOSTI ZVONCA 0x7fff...018 zvonce 
    //munmap((void*)0x7fffe2fdf000, 0x7fffe2fe0000-0x7fffe2fdf000); // SEG 0x7fffe2fdf000 seg na hipMemcpyHtoD 
    //munmap((void*)0x7fffe2fe7000, 0x7fffe2fe8000-0x7fffe2fe7000); // SEG 0x7fffe2fe7000 seg na hipMemcpyHtoD
    //munmap((void*)0x7fffe2feb000, 0x7fffe2fec000-0x7fffe2feb000); // SEG 0x7fffe2feb000 seg na hipMemcpyHtoD
    //munmap((void*)0x7fffe2fef000, 0x7fffe2ff0000-0x7fffe2fef000); // SEG 0x7fffe2fef000 seg na hipMemcpyHtoD
    munmap((void*)0x7fffce400000, 0x7fffce600000-0x7fffce400000);
    munmap((void*)0x7fffcf000000, 0x7fffcf200000-0x7fffcf000000);
    munmap((void*)0x7fffcf400000, 0x7fffcf577000-0x7fffcf400000);
    munmap((void*)0x7fffe2fe0000, 0x7fffe2fe1000-0x7fffe2fe0000);
    munmap((void*)0x7fffe2fe1000, 0x7fffe2fe2000-0x7fffe2fe1000);
    munmap((void*)0x7fffe2fe2000, 0x7fffe2fe3000-0x7fffe2fe2000);
    munmap((void*)0x7fffe2fe3000, 0x7fffe2fe4000-0x7fffe2fe3000);
    munmap((void*)0x7fffe2fe4000, 0x7fffe2fe5000-0x7fffe2fe4000);
    munmap((void*)0x7fffe2fe5000, 0x7fffe2fe6000-0x7fffe2fe5000);
    munmap((void*)0x7fffe2fe6000, 0x7fffe2fe7000-0x7fffe2fe6000);
    munmap((void*)0x7fffe2fe8000, 0x7fffe2fe9000-0x7fffe2fe8000);
    munmap((void*)0x7fffe2fe9000, 0x7fffe2fea000-0x7fffe2fe9000);
    munmap((void*)0x7fffe2fea000, 0x7fffe2feb000-0x7fffe2fea000);
    munmap((void*)0x7fffe2fec000, 0x7fffe2fed000-0x7fffe2fec000);
    munmap((void*)0x7fffe2fed000, 0x7fffe2fee000-0x7fffe2fed000);
    munmap((void*)0x7fffe2fee000, 0x7fffe2fef000-0x7fffe2fee000);
    munmap((void*)0x7fffe2ff0000, 0x7fffe2ff1000-0x7fffe2ff0000);
    munmap((void*)0x7fffe2ff1000, 0x7fffe2ff2000-0x7fffe2ff1000);
    munmap((void*)0x7fffe2ff2000, 0x7fffe2ff3000-0x7fffe2ff2000);
    //map(getpid());
    /*
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);
    */

    //dev/nvidia0
    //dev/nvidia-uvm
    munmap((void*)0x205000000 , 0x205200000-0x205000000);   // ne

    //dev/nvidiactl 
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x200000000, 0x200200000-0x200000000);
    munmap((void*)0x204c00000, 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000, 0x205000000-0x204e00000);   // ne
    munmap((void*)0x205200000, 0x205400000-0x205200000);   // ne

    /*hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);*/


    //memset((void*)c , 0x0 ,512);
    
    clear_nvctrl();
    printf("*************cuCopyHosttoDevice*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int)*N);

    /*
###

    p $_siginfo._sifields._sigfault.si_addr

    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    printf("*************cuda_memcpyHtod_1*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    hipMemcpyHtoD(d_b, b, sizeof(int) * N);

    printf("_____%llx_____  \n" ,d_a);
    printf("_____%llx_____  \n" ,d_b);
    printf("_____%llx_____  \n" ,d_c);

    munmap((void*)0x203c00000 , 0x204a00000-0x203c00000); // nista
    munmap((void*)0x204c10000 , 0x204c10000-0x204a00000); // nista
    munmap((void*)0x205400000 , 0x205600000-0x205400000); // nista
    munmap((void*)0x205800000 , 0x205a00000-0x205800000); // nista

    clear_nvctrl();
    void *args[3] = {&d_a, &d_b, &d_c};
    
    printf("*************cuda_LacunhKernel*************\n");
    hipModuleLaunchKernel(function, 32, 16, 1, 1, 1, 1, 0, 0, args, 0);
    //dump_small((void*)0x200400000 , (void*)0x203c00000); 
###
    */
    
    //dump_small((void*)0x200400000 , (void*)0x203c00000);

    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_a, sizeof(int) * N);
    hexdump((void*)c , 0x20);

    // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
    
    //printf("CONTEXXXXXXXXXXXXX FREEEEEEEEEEEEEEEEEEEEEEEEEEEEEe\n");
    //hipCtxDestroy(context);
    return 0;
}
// p $_siginfo._sifields._sigfault.si_addr

/*
0x7fffe2feb010 : 1550 
0x7fffe2feb014 : 74e4c9c0 
0x7fffe2feb018 : 90017        #door bell value
0x7fffe2feb01c : ffff0000 

*/