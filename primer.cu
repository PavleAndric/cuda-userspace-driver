#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"

#define N 512
int main()
{   
    int a[N], b[N],control[N];
    int c[N];
    hipDeviceptr_t d_a, d_b, d_c;
    for (int i = 0; i < N; ++i)
    {
        a[i] = i;
        b[i] = i;
        control[i] = i + i;
    }

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");
    //uint32_t *door =(uint32_t *)0x204a0fff0;

    printf("*************cuda_device_get*************\n");
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    printf("*************cuda_create_contex*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
      
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    const char *module_file = "kernel.ptx";
    hipModuleLoad(&module, module_file);

    hipFunction_t function;
    const char *kernel_name = "kernel";
    hipModuleGetFunction(&function, module, kernel_name);

    //dev/nvidia0
    //dev/nvidia-uvm
    munmap((void*)0x205000000 , 0x205200000-0x205000000);   // ne

    //dev/nvidiactl 
    //munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
    //munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
    //munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    munmap((void*)0x204c00000 , 0x204e00000-0x204c00000);   // ne
    munmap((void*)0x204e00000 , 0x205000000-0x204e00000);   // ne
    munmap((void*)0x205200000 , 0x205400000-0x205200000);   // ne

		mprotect((void*)0x200000000, 0x200200000-0x200000000, PROT_READ );


    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    //printf("DESINGER %llx %p\n" , d_a , &d_a);
    printf("*************cuda_memcpyHtod_1*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int) * N);
    printf("*************cuda_memcpyHtod_2*************\n");
    hipMemcpyHtoD(d_b, b, sizeof(int) * N);

    //unsigned int * romcina = (unsigned int *)0xB0000B;
    printf("_____%llx_____  \n" ,d_a);
    printf("_____%llx_____  \n" ,d_b);
    printf("_____%llx_____  \n" ,d_c);
    void *args[3] = {&d_a, &d_b, &d_c};
  
    //munmap((void*)0x200436000 , 0x202c00000-0x200436000); 
    //munmap((void*)0x200400000 , 0x200435000-0x200400000); 
    //munmap((void*)0x200230000, 0x200400000-0x200230000);

    munmap((void*)0x203c00000 , 0x204a00000-0x203c00000); // nista
    munmap((void*)0x204c10000 , 0x204c10000-0x204a00000); // nista
    munmap((void*)0x205400000 , 0x205600000-0x205400000); // nista
    munmap((void*)0x205800000 , 0x205a00000-0x205800000); // nista

    uint64_t region = gas(getpid()) + 0x177000 ;
    
    /*mprotect((void*)region, region+0xa89000, PROT_READ | PROT_WRITE);
    uint64_t *p_addr = (uint64_t*)((region & 0xFFFFFF000000) | 0xe3b300); p_addr = NULL; 
    uint64_t *ne_znam = (uint64_t*)((region & 0xFFFFFF000000) | 0xfffdc0);
    printf("IDE_GAS %p %x \n" ,ne_znam  , *ne_znam);
    uint32_t *ptr = (uint32_t*)0x205607f7c; // insteresantan ? 
    // map(getpid());*/
    
    memset((void*)0x200400000 ,0x0 , 0x203c00000-0x200400000);

    printf("*************cuda_LacunhKernel*************\n");
    hipModuleLaunchKernel(function, 32, 16, 1, 1, 1, 1, 0, 0, args, 0);
    dump_small((void*)0x200400000 , (void*)0x203c00000); // 0x20043529c

    //dump_small((void*)0x7ffff7fab000 , (void*)0x203c00000); // 0x20043529c

    printf("*************cuda_memcpyDtoh*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);
    for(int i = 0 ; i < N ; i ++){ assert(c[i] == control[i]); }

    /* // Free device memory
    printf("*************cuda_Free_1*************\n");
    hipFree(d_a);
    printf("*************cuda_Free_2*************\n");
    hipFree(d_b);
    printf("*************cuda_Free_3*************\n");
    hipFree(d_c);
		printf(":D :D :D\n"); */
    
    return 0;
}
// p $_siginfo._sifields._sigfault.si_addr

// 0x200200348 , 0x200200348 4350c4 , 4350c4 , 4350c4
// 0x20020208c
// 0x200224008
// 0x200224008

// 0x204a0fff0

/*
0x7fa6f5e3b300

7fa6f2000000-7fa6f4400000 ---p 00000000 00:00 0 
7fa6f4400000-7fa6f4600000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f4600000-7fa6f4800000 rw-s 00000000 00:01 166935                     /dev/zero (deleted)
7fa6f4800000-7fa6f4a00000 rw-s 00000000 00:01 166936                     /dev/zero (deleted)
7fa6f4a00000-7fa6f5000000 ---p 00000000 00:00 0 
7fa6f5000000-7fa6f5200000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f5200000-7fa6f5400000 rw-s 00000000 00:01 166939                     /dev/zero (deleted)
7fa6f5400000-7fa6f5577000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f5577000-7fa6f6000000 ---p 00000000 00:00 0  //////////////////////////// OVDE  JE  0x7fa6f5e3b300
7fa6f6fbc000-7fa6f6fbd000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fbd000-7fa6f6fbe000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fbe000-7fa6f6fbf000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fbf000-7fa6f6fc0000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc0000-7fa6f6fc1000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc1000-7fa6f6fc2000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc2000-7fa6f6fc3000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc3000-7fa6f6fc4000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc4000-7fa6f6fc5000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc5000-7fa6f6fc6000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc6000-7fa6f6fc7000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc7000-7fa6f6fc8000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc8000-7fa6f6fc9000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fc9000-7fa6f6fca000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fca000-7fa6f6fcb000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fcb000-7fa6f6fcc000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fcc000-7fa6f6fcd000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fcd000-7fa6f6fce000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fce000-7fa6f6fcf000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fcf000-7fa6f6fd0000 rw-s 00000000 00:05 1006                       /dev/nvidiactl
7fa6f6fd0000-7fa6f7fde000 ---p 00000000 00:00 0 
7fa6f7fde000-7fa6f87de000 rw-p 00000000 00:00 0 
7fa6f87de000-7fa6f87df000 ---p 00000000 00:00 0 
7fa6f87df000-7fa6f8fdf000 rw-p 00000000 00:00 0 
7fa6f8fdf000-7fa700000000 ---p 00000000 00:00 0


*/
