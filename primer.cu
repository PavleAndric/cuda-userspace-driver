#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"

#define N 32
int main()
{   
    int a[N], b[N],c[N],control[N];
    hipDeviceptr_t d_a, d_b, d_c;
    hipFunction_t function;
    hipModule_t module;

    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i;}

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");
    mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);

    hipDevice_t device;
    hipDeviceGet(&device , 0);
    printf("DEVICE = %p -> %x \n" , &device , device);

		printf("*************cuda_create_contex_start*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    printf("*************cuda_create_contex_ende*************\n");
    mprotect((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000 , PROT_READ | PROT_WRITE);

    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);

    clear_nvctrl(); memset((void*)0x200200000 , 0x0 , 0x200000); 
    printf("*************cuda_module_load*************\n");
    int res_1 = hipModuleLoad(&module, "kernel.ptx");
    assert(res_1 == 0);

    int res_2 = hipModuleGetFunction(&function, module, "kernel");
    assert(res_2 == 0);

    printf("*************cuCopyHosttoDevice*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int)*N);
    printf("*************cuCopyHosttoDevice*************\n");
    hipMemcpyHtoD(d_b, a, sizeof(int)*N);

    printf("*************KERNEL_LAUNNCH*************\n");
    void *args[3] = { &d_a, &d_b, &d_c};
    clear_nvctrl(); memset((void*)0x200200000 , 0x0 , 0x200000); 

    int res_3 = hipModuleLaunchKernel(function, 1, 1, 1, N, 1, 1, 0, 0, args, 0);
    assert(res_3 == 0);

    dump_small((void*)0x200400000 ,(void*)0x203c00000);
    for(uint32_t *ptr = (uint32_t*)0x200200000 ; ptr <(uint32_t*)0x200400000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}


    printf("*************DEVICE_TO_HOST*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);
    for(int i  = 0 ; i < N ; i++){printf("%x " ,c[i]);}printf("\n");

    return 0;
}

//for(uint32_t *ptr = (uint32_t*)0x200400000 ; ptr <(uint32_t*)0x203c00000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}
//printf("NV_0\n");
//for(uint32_t *ptr = (uint32_t*)0x200200000 ; ptr <(uint32_t*)0x200400000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}

//munmap((void*)0x200000000, 0x200200000-0x200000000);
//munmap((void*)0x205200000, 0x205400000-0x205200000);   // ne
//munmap((void*)0x205000000 , 0x205200000-0x205000000);  // ne
//munmap((void*)0x204c00000, 0x204e00000-0x204c00000);   // ne
//munmap((void*)0x204e00000, 0x205000000-0x204e00000);   // ne



//dump_small((void*)0x200400000 ,(void*)0x203c00000);
//for(uint32_t *ptr = (uint32_t*)0x200200000 ; ptr <(uint32_t*)0x200400000 ; ptr ++){ if(*ptr){printf("%p: %x\n " , ptr , *ptr);}}
//clear_nvctrl(); memset((void*)0x200200000 , 0x0 , 0x200000); 


//munmap((void*)0x200400000 , 0x203c00000-0x200400000);   // mora
//munmap((void*)0x204a00000 , 0x204c00000-0x204a00000);   // mora 
//munmap((void*)0x205600000 , 0x205800000-0x205600000);   // mora
    