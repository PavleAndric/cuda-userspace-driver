#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <assert.h>
#include "helpers.h"

#define N 10
int main()
{   
    int a[N], b[N],c[N],control[N];
    hipDeviceptr_t d_a, d_b, d_c , d_d;

    for (int i = 0; i < N; ++i){a[i] = i; b[i] = i;}

    printf("*************cuda_init*************\n");
    hipInit(0);
    printf("*************end_cudainit*************\n");
    mprotect((void*)0x200000000 , 0x300200000-0x200000000 , PROT_READ | PROT_WRITE);

    hipDevice_t device;
    hipDeviceGet(&device , 0);
    printf("DEVICE = %p -> %x \n" , &device , device);

		printf("*************cuda_create_contex_start*************\n");
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    printf("*************cuda_create_contex_ende*************\n");
    mprotect((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000 , PROT_READ | PROT_WRITE);


    printf("*************cuda_malloc_1*************\n");
    hipMalloc(&d_a, sizeof(int) * N);
    printf("*************cuda_malloc_2*************\n");
    hipMalloc(&d_b, sizeof(int) * N);
    printf("*************cuda_malloc_3*************\n");
    hipMalloc(&d_c, sizeof(int) * N);
    uint64_t *door_bell = (uint64_t*)0x7ffff7fab090;
    hipMalloc(&d_d, sizeof(int) * 0x1000000);

    map(getpid());
    exit(1);

    printf("*************cuCopyHosttoDevice*************\n");
    hipMemcpyHtoD(d_a, a, sizeof(int)*N);
    hipMemcpyHtoD(d_b, b, sizeof(int)*N);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////
    printf("*************cuda_module_load*************\n");
    hipModule_t module;
    hipFunction_t function;
    printf("da = %lx db = %lx dc = %lx\n" ,(uint64_t)d_a , (uint64_t)d_b ,(uint64_t)d_c);

    hipMemcpyDtoH(b, d_a, sizeof(int) * N);

    int res = hipModuleLoad(&module, "kernel.cubin");
    printf("res = %x\n" , res);
    assert(res == 0);

    munmap((void*)0x7fffcf577000 , 0x7fffe0000000-0x7fffcf577000);
    munmap((void*)0x7fffcc000000 , 0x7fffce400000-0x7fffcc000000);
    
    printf("*************FUNCTION*************\n");
    int res_2 = hipModuleGetFunction(&function, module, "kernel"); assert(res_2 == 0);  // p $_siginfo._sifields._sigfault.si_addr
    clear_nvctrl();
    munmap((void*)205600000 , 205800000-205600000);

    printf("*************KERNEL_LAUNNCH*************\n");
    void *args[3] = { &d_a, &d_a, &d_c};
    int res_3 = hipModuleLaunchKernel(function, 1, 1, 1, N, 1, 1, 0, 0, args, 0);
    assert(res_3 == 0);

    printf("*************DEVICE_TO_HOST*************\n");
    hipMemcpyDtoH(c, d_c, sizeof(int) * N);

    return 0;
}

