
#include <hip/hip_runtime.h>
extern "C" __global__ void kernel(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    c[tid] = a[tid] + b[tid];
}