
#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" __global__ void kernel(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    c[tid] = a[tid] + b[tid];
}