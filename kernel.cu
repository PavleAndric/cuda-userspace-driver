
#include <hip/hip_runtime.h>
extern "C" __global__ void kernel(int *a, int *b, int *c)
{
    int tidX = blockIdx.x;    // X dimension of the grid
    int tidY = blockIdx.y;    // Y dimension of the grid
    int tid = tidY * blockDim.x * gridDim.x + tidX;  // Calculate the unique thread ID

    c[tid] = a[tid] + b[tid];
}